#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <time.h> 
#include <windows.h>
#include <d2d1.h>
#include <d2d1helper.h>
#pragma comment(lib, "d2d1")
#include <mmsystem.h>
#pragma comment(lib, "winmm.lib")
#include "pegazus_main.h"
#include "PEGA_FORMAT_OBJ_NEW.h"
#include "PEGA_FORMAT_BMP.h"

//***********STANDARD WIN32API WINDOWS HANDLING ************
HINSTANCE hInstGlob;
int SajatiCmdShow;
HWND Form1; // Windows handler
LRESULT CALLBACK WndProc0(HWND, UINT, WPARAM, LPARAM);
//******************************************************

//************************
void PEGA_drawing3D_update(void);
//************************************

//*********************************
// Entry point of the program
//*********************************
int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, PSTR szCmdLine, int iCmdShow)
{
 MSG msg;
 WNDCLASS wndclass0;
 SajatiCmdShow = iCmdShow;
 hInstGlob = hInstance;

 //*********************************
 // Preparing the Window class instance
 //*********************************
 wndclass0.style = CS_HREDRAW | CS_VREDRAW;
 wndclass0.lpfnWndProc = WndProc0;
 wndclass0.cbClsExtra = 0;
 wndclass0.cbWndExtra = 0;
 wndclass0.hInstance = hInstance;
 wndclass0.hIcon = LoadIcon(NULL, IDI_APPLICATION);
 wndclass0.hCursor = LoadCursor(NULL, IDC_ARROW);
 wndclass0.hbrBackground = (HBRUSH)GetStockObject(LTGRAY_BRUSH);
 wndclass0.lpszMenuName = NULL;
 wndclass0.lpszClassName = TEXT("WIN0");

 //*********************************
 // Registering the window class
 //*********************************
 if (!RegisterClass(&wndclass0))
 {
  MessageBox(NULL, TEXT("Error:Program initialisation process."), TEXT("Program Start"), MB_ICONERROR);
  return 0;
 }

 //*********************************
 // Creating the window
 //*********************************
 Form1 = CreateWindow(TEXT("WIN0"),
  TEXT("CUDA - DIRECT2D"),
  (WS_OVERLAPPED | WS_SYSMENU | WS_THICKFRAME | WS_MAXIMIZEBOX | WS_MINIMIZEBOX),
  0,
  0,
  SCREEN_WIDTH,
  SCREEN_HEIGHT,
  NULL,
  NULL,
  hInstance,
  NULL);

 //*********************************
 // Showing the window
 //*********************************
 ShowWindow(Form1, SajatiCmdShow);
 UpdateWindow(Form1);

 //*********************************
 // Window message handling
 //*********************************
 while (GetMessage(&msg, NULL, 0, 0))
 {
  TranslateMessage(&msg);
  DispatchMessage(&msg);
 }
 return msg.wParam;
}

//*********************************
// The window’s callback function: event handling
//*********************************
LRESULT CALLBACK WndProc0(HWND hwnd, UINT message, WPARAM wParam, LPARAM lParam)
{
 HDC hdc;
 PAINTSTRUCT ps;
 FILE* myfile;
 unsigned int xPos, yPos, xPos2, yPos2, fwButtons;

 switch (message)
 {
  //*********************************
  // Creating the window
  //*********************************
 case WM_CREATE:
  /*Init*/;
  if ((joyGetNumDevs()) > 0) joySetCapture(hwnd, JOYSTICKID1, NULL, FALSE);
  fopen_s(&myfile, "CUDA_benchmark.txt", "wt");
  fclose(myfile);
  PEGA_benchmark_start();
  PEGA_init(hwnd);
  PEGA_init_3D();
  PEGA_benchmark_stop("----------->>>>init");

  PEGA_benchmark_start();
  PEGA_create_HOST_CUDA_3D_vertex_list(10000000);  

  PEGA_benchmark_stop("memory allocation");
  PEGA_drawing3D();
  PEGA_3D_vertex_reset();
  PEGA_obj_loader("3dmodel.obj");
  PEGA_log_dev_status(raw_vertices_length,MAX_3d_vertex_count);
  PEGA_push_3D_triangles_to_GPU();
  return 0;
  //*********************************
  // To prevent flickering
  //*********************************
 case WM_ERASEBKGND:
  return (LRESULT)1;
 case MM_JOY1MOVE:
  fwButtons = wParam;
  xPos = LOWORD(lParam);
  yPos = HIWORD(lParam);
   if (xPos == 65535) {
    rot_deg2.y += 5.0; PEGA_drawing3D_update();
   }
   else if (xPos == 0) {
    rot_deg2.y -= 5.0; PEGA_drawing3D_update();
   }
   if (yPos == 65535) {
    rot_deg2.x += 5.0; PEGA_drawing3D_update();
   }
   else if (yPos == 0) {
    rot_deg2.x -= 5.0; PEGA_drawing3D_update();
   }
   if (fwButtons == 128) {
    rot_deg2.z += 5.0; PEGA_drawing3D_update();
   }
   else if (fwButtons == 64) {
    rot_deg2.z -= 5.0; PEGA_drawing3D_update();
   }
   if (rot_deg2.y > 359) {
    rot_deg2.y = 0; PEGA_drawing3D_update();
   }
   else if (rot_deg2.y < 0) {
    rot_deg2.y = 358; PEGA_drawing3D_update();
   }
   if (rot_deg2.x > 359) {
    rot_deg2.x = 0; PEGA_drawing3D_update();
   }
   else if (rot_deg2.x < 0) {
    rot_deg2.x = 358; PEGA_drawing3D_update();
   }
   if (rot_deg2.z > 359) {
    rot_deg2.z = 0; PEGA_drawing3D_update();
   }
   else if (rot_deg2.z < 0) {
    rot_deg2.z = 358; PEGA_drawing3D_update();
   }
  if (fwButtons == 2)
  {
   zoom_value *= 1.02;
   PEGA_zoom_in();
   PEGA_drawing3D_update();
  }
  else if (fwButtons == 4)
  {
   zoom_value /= 1.02;
   PEGA_zoom_out();
   PEGA_drawing3D_update();
  }
  break;
  //*********************************
  // Drawing the window’s client area
  //*********************************
 case WM_PAINT:
  hdc = BeginPaint(hwnd, &ps);
  EndPaint(hwnd, &ps);
  return 0;
  //*********************************
  // Closing the window
  //*********************************
 case WM_CLOSE:
  PEGA_benchmark_start();
  PEGA_free_Direct2D();
  PEGA_free3D();
  PEGA_benchmark_stop("----------->>>>memory free up");
  DestroyWindow(hwnd);
  return 0;
  //*********************************
  // Destroying the window
  //*********************************
 case WM_DESTROY:
  PostQuitMessage(0);
  return 0;
 }
 return DefWindowProc(hwnd, message, wParam, lParam);
}

void PEGA_drawing3D_update(void)
{
 char tempstr2[128], tempstr[128];

 PEGA_start_fps_benchmark();
 PEGA_benchmark_start();
 PEGA_clearscreen();
 PEGA_benchmark_stop("----------->>>>clearscreen");

 PEGA_benchmark_start();
 PEGA_rotate_3D();
 PEGA_benchmark_stop("3D rotation");

 PEGA_benchmark_start();
 PEGA_render_3D();
 PEGA_benchmark_stop("rendering");

 char errmessage[256];
 strcpy_s(errmessage, hipGetErrorString(hipGetLastError()));
 SetWindowTextA(Form1, errmessage);

 PEGA_benchmark_start();
 if (CUDA_DEVICE_COUNT > 1)
 {
  PEGA_merge_down_zbuffers();  
 }
 PEGA_swap_buffer();
 PEGA_benchmark_stop("swap buffer");
 PEGA_get_fps_benchmark();

 strcpy(tempstr2, "FPS: "); _itoa(fps_stat, tempstr, 10); strcat(tempstr2, tempstr);
 SetWindowTextA(Form1, tempstr2);
}
