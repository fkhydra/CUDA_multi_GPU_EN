#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <time.h> 
#include <windows.h>
#include <d2d1.h>
#include <d2d1helper.h>
#pragma comment(lib, "d2d1")
#include <mmsystem.h>
#pragma comment(lib, "winmm.lib")
#include "pegazus_main.h"
#include "PEGA_FORMAT_OBJ_NEW.h"

//***********STANDARD WIN32API WINDOWS HANDLING************
HINSTANCE hInstGlob;
int SajatiCmdShow;
HWND Form1; //Windows handler
LRESULT CALLBACK WndProc0(HWND, UINT, WPARAM, LPARAM);
//******************************************************

//************************
void PEGA_drawing2D(void);
//************************************

//*********************************
//Entry point of the program
//*********************************
int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, PSTR szCmdLine, int iCmdShow)
{
 MSG msg;
 WNDCLASS wndclass0;
 SajatiCmdShow = iCmdShow;
 hInstGlob = hInstance;

 //*********************************
 //Preparing the Window class instance
 //*********************************
 wndclass0.style = CS_HREDRAW | CS_VREDRAW;
 wndclass0.lpfnWndProc = WndProc0;
 wndclass0.cbClsExtra = 0;
 wndclass0.cbWndExtra = 0;
 wndclass0.hInstance = hInstance;
 wndclass0.hIcon = LoadIcon(NULL, IDI_APPLICATION);
 wndclass0.hCursor = LoadCursor(NULL, IDC_ARROW);
 wndclass0.hbrBackground = (HBRUSH)GetStockObject(LTGRAY_BRUSH);
 wndclass0.lpszMenuName = NULL;
 wndclass0.lpszClassName = TEXT("WIN0");

 //*********************************
 //Registering the window class
 //*********************************
 if (!RegisterClass(&wndclass0))
 {
  MessageBox(NULL, TEXT("Error:Program initialisation process."), TEXT("Program Start"), MB_ICONERROR);
  return 0;
 }

 //*********************************
 //Creating the window
 //*********************************
 Form1 = CreateWindow(TEXT("WIN0"),
  TEXT("CUDA - DIRECT2D"),
  (WS_OVERLAPPED | WS_SYSMENU | WS_THICKFRAME | WS_MAXIMIZEBOX | WS_MINIMIZEBOX),
  0,
  0,
  SCREEN_WIDTH,
  SCREEN_HEIGHT,
  NULL,
  NULL,
  hInstance,
  NULL);

 //*********************************
 //Showing the window
 //*********************************
 ShowWindow(Form1, SajatiCmdShow);
 UpdateWindow(Form1);

 //*********************************
 //Window message handling
 //*********************************
 while (GetMessage(&msg, NULL, 0, 0))
 {
  TranslateMessage(&msg);
  DispatchMessage(&msg);
 }
 return msg.wParam;
}

//*********************************
//The window’s callback function: event handling
//*********************************
LRESULT CALLBACK WndProc0(HWND hwnd, UINT message, WPARAM wParam, LPARAM lParam)
{
 HDC hdc;
 PAINTSTRUCT ps;

 switch (message)
 {
  //*********************************
  //Creating the window
  //*********************************
 case WM_CREATE:
  /*Init*/;
  srand((unsigned)time(NULL));
  PEGA_init(hwnd);
  PEGA_create_HOST_2D_point_list(10000);
  PEGA_create_CUDA_2D_point_list(10000);
  PEGA_create_HOST_2D_line_list(10000);
  PEGA_create_CUDA_2D_line_list(10000);
  PEGA_create_HOST_2D_triangle_list(10000);
  PEGA_create_CUDA_2D_triangle_list(10000);
  return 0;
  //*********************************
  //To prevent flickering
  //*********************************
 case WM_ERASEBKGND:
  return (LRESULT)1;
  //*********************************
  //Drawing the window’s client area
  //*********************************
 case WM_PAINT:
  hdc = BeginPaint(hwnd, &ps);
  EndPaint(hwnd, &ps);
  PEGA_drawing2D();
  return 0;
  //*********************************
  //Closing the window
  //*********************************
 case WM_CLOSE:
  PEGA_free_Direct2D();
  PEGA_free2D();
  DestroyWindow(hwnd);
  return 0;
  //*********************************
  //Destroying the window
  //*********************************
 case WM_DESTROY:
  PostQuitMessage(0);
  return 0;
 }
 return DefWindowProc(hwnd, message, wParam, lParam);
}

void PEGA_drawing2D(void)
{
 int i, j;
 PEGA_clearscreen();

 // drawing points
 PEGA_2D_point_reset();
 for (j = 0; j < 5000; j += 1)
  PEGA_add_2D_point(get_rnd(1899), get_rnd(999), RGB(255, 0, 0));

 // drawing lines
 PEGA_2D_line_reset();
 PEGA_add_2D_line(0, 0, 1900,1000,RGB(0, 0, 0));
 PEGA_add_2D_line(1900, 0, 0, 1000, RGB(0, 0, 0));
 for (j = 0; j < 1900; j += 100)
  PEGA_add_2D_line(j, 0, j, 1000, RGB(0, 0, 0));
 PEGA_add_2D_line(0, 2, 1898, 2, RGB(0, 0, 0));
 PEGA_add_2D_line(1898, 2, 1898, 998, RGB(0, 0, 0));
 PEGA_add_2D_line(0, 998, 0, 2, RGB(0, 0, 0));
 PEGA_add_2D_line(1898, 998, 0, 998, RGB(0, 0, 0));

 // drawing triangles
 PEGA_2D_triangle_reset();
 PEGA_add_2D_triangle(get_rnd(1899), get_rnd(999), get_rnd(1899), get_rnd(999), get_rnd(1899), get_rnd(999), RGB(get_rnd(255), get_rnd(255), get_rnd(255)));

 PEGA_push_points_to_GPU();
 PEGA_push_lines_to_GPU();
 PEGA_push_triangles_to_GPU();  
 PEGA_render_2D(); 
 
 PEGA_merge_down_2D_buffer();
 PEGA_swap_buffer();

 char errmessage[256];
 strcpy_s(errmessage, hipGetErrorString(hipGetLastError()));
 SetWindowTextA(Form1, errmessage);
}
